#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void init(int *a, int alen) {
    unsigned id = threadIdx.x;
    if(id<alen) a[id] = 0;
}
 int main(){
    int *da, N;
    N=32;
    hipMalloc(&da, N * sizeof(int));
    
    init<<<1, N>>>(da, N);
    hipDeviceSynchronize();

    int a[N];
    hipMemcpy(a, da, N* sizeof(int), hipMemcpyDeviceToHost);
    for(int i=0;i<N;++i){
        printf("%d ",a[i]);
    }
    printf("\n");
    return 0;
 }
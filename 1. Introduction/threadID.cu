#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 100

__global__ void dkernel() {
    printf("%d\n",threadIdx.x);
}

int main(){
    dkernel<<<1,N>>>();
    hipDeviceSynchronize();
    return 0;
}
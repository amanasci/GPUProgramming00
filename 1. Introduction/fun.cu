#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 100


__global__ void fun() {
    for(int i=0;i<N;++i){
        printf("%d\n",i*i)
    }
}

int main(){
    fun<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}